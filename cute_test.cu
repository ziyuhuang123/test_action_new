#include "hip/hip_runtime.h"
#include <cute/tensor.hpp>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

// z = ax + by + c
template <int kNumElemPerThread = 8>
__global__ void vector_add_local_tile_multi_elem_per_thread_half(
    half *z, int num, const half *x, const half *y, const half a, const half b, const half c) {
  using namespace cute;

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= num / kNumElemPerThread) { // 未处理非对齐问题
    return;
  }

  Tensor tz = make_tensor(make_gmem_ptr(z), make_shape(num));
  Tensor tx = make_tensor(make_gmem_ptr(x), make_shape(num));
  Tensor ty = make_tensor(make_gmem_ptr(y), make_shape(num));

  Tensor tzr = local_tile(tz, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor txr = local_tile(tx, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor tyr = local_tile(ty, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));

  Tensor txR = make_tensor_like(txr);
  Tensor tyR = make_tensor_like(tyr);
  Tensor tzR = make_tensor_like(tzr);

  // LDG.128
  copy(txr, txR);
  copy(tyr, tyR);

  half2 a2 = {a, a};
  half2 b2 = {b, b};
  half2 c2 = {c, c};

  auto tzR2 = recast<half2>(tzR);
  auto txR2 = recast<half2>(txR);
  auto tyR2 = recast<half2>(tyR);

#pragma unroll
  for (int i = 0; i < size(tzR2); ++i) {
    // two hfma2 instruction
    tzR2(i) = txR2(i) * a2 + (tyR2(i) * b2 + c2);
  }

  auto tzRx = recast<half>(tzR2);

  // STG.128
  copy(tzRx, tzr);
}

int main() {
    const int kNumElemPerThread = 8; // 定义此常量

    // 定义向量大小
    int num = 1024; // 示例大小

    // 为向量分配内存
    half *x, *y, *z;
    hipMallocManaged(&x, num * sizeof(half));
    hipMallocManaged(&y, num * sizeof(half));
    hipMallocManaged(&z, num * sizeof(half));

    // 初始化向量 x 和 y
    for (int i = 0; i < num; ++i) {
        x[i] = half(i);  // 示例数据
        y[i] = half(num - i);  // 示例数据
    }

    // 定义操作的系数
    half a = half(1.0), b = half(2.0), c = half(3.0);

    // 定义 kernel 的执行配置
    int threadsPerBlock = 256;
    int elementsPerBlock = kNumElemPerThread * threadsPerBlock;
    int numBlocks = (num + elementsPerBlock - 1) / elementsPerBlock;

    // 调用 kernel
    vector_add_local_tile_multi_elem_per_thread_half<kNumElemPerThread><<<numBlocks, threadsPerBlock>>>(
        z, num, x, y, a, b, c);
    hipDeviceSynchronize();  // 等待 GPU 完成

    // 验证结果（可选）
    bool valid = true;
    for (int i = 0; i < num; ++i) {
        if (static_cast<float>(z[i]) != static_cast<float>(a) * static_cast<float>(x[i]) + static_cast<float>(b) * static_cast<float>(y[i]) + static_cast<float>(c)) {
            std::cout << "Mismatch at " << i << ": " << static_cast<float>(z[i]) << " != " << (static_cast<float>(a) * static_cast<float>(x[i]) + static_cast<float>(b) * static_cast<float>(y[i]) + static_cast<float>(c)) << std::endl;
            valid = false;
            break;
        }
    }
    if (valid) {
        std::cout << "Results are correct!" << std::endl;
    }

    // 清理
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
